#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES // for use of M_PI
#include "runge_kuttaCUDA.cuh"
#include "runge_kutta.h" // used for rkCalc()
#include "../Thrust_Files/acceleration.h" //used for calc_accel() and calc_coast()
#include "rkParameters.h" // the struct containing the values passed to rk4simple()
#include "../Earth_calculations/orbitalMotion.h"
#include "../Genetic_Algorithm/geneticAlgorithm.h" // selectWinners()
#include "../Genetic_Algorithm/ga_crossover.h" // crossover()
#include "../Genetic_Algorithm/gaConstants.h" // SURVIVOR_COUNT
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file
#include <chrono>
#include <algorithm> // sort(), shuffle()
#include <random>

// THIS FUNCTION HAS NOT BEEN COMPLETED AND IS NOT IN USE
Individual bestChange(Individual original, double timeInitial, double stepSize, double absTol){
    Individual best = original;
    Individual cur = original;

    elements<double> output;

    double parameterChange;
    
    // get the original result
    best.startParams.parametersRK4Simple(timeInitial, stepSize, absTol, output);
    best.finalPos = output;
    best.posDiff =  sqrt(pow(R_FIN_AST - output.r, 2) + pow(THETA_FIN_AST - fmod(output.theta, 2 * M_PI), 2) + pow(Z_FIN_AST - output.z, 2));
    best.velDiff =  sqrt(pow(VR_FIN_AST - output.vr, 2) + pow(VTHETA_FIN_AST - output.vtheta, 2) + pow(VZ_FIN_AST - output.vz, 2));

    // get results for each changed variable
    // gamma
    parameterChange = 0.1;
    for(int i = 0; i < 7; i++){
        cur.startParams.coeff.gamma[i] += parameterChange;
        cur.startParams.parametersRK4Simple(timeInitial, stepSize, absTol, output);
        if(!greaterInd(best, cur)){
            best = cur;
        }
        cur.startParams.coeff.gamma[i] -= parameterChange;
    }
    //tau
    parameterChange = 0.1;
    for(int i = 0; i < 3; i++){
        cur.startParams.coeff.tau[i] += parameterChange;
        cur.startParams.parametersRK4Simple(timeInitial, stepSize, absTol, output);
        if(!greaterInd(best, cur)){
            best = cur;
        }
        cur.startParams.coeff.tau[i] -= parameterChange;
    }
    //coast

    return best;
}

double optimize(const int numThreads, const int blockThreads){
    double calcPerS = 0;
    time_t timeSeed = time(0);
    std::cout << "Time seed for this run: " << timeSeed << std::endl; // note there are other mt_rands in the code that use different seeds
    std::cout << "------------------------------------------------------------------------" << std::endl;
    std::mt19937_64 mt_rand(timeSeed);

     // input parameters for rk4Simple which are the same for each thread
    double timeInitial = 0; // the starting time of the trip is always defined as zero   
    double absTol = RK_TOL; // the tolerance is a constant number that is shared amongst all runs
    double stepSize = (orbitalPeriod - timeInitial) / MAX_NUMSTEPS; // the starting step size- same for each run- note that the current step size varies throughout each run


    Individual *inputParameters = new Individual[numThreads]; // contains all input parameters besides those which are always common amongst every thread

    const int numStarts = 14; // the number of different sets of starting parameters in the input file

    std::ifstream starts;
    starts.open("../optimizedVector.bin", std::ifstream::in|std::ios::binary); // a file containing the final parameters of converged results from CPU calculations

    double startDoubles;

    // sort the data into 2 dimensions
    // one row is one set of starting parameters
    // each column is a specific variable:
    //    0-6 gamma
    //    7-9 tau
    //    10-12 launch angles
    //    13 trip time
    //    14-19 coast
    double arrayCPU[numStarts][OPTIM_VARS];
    for(int i = 0; i < OPTIM_VARS; i++){ // rows
        for(int j = 0; j < numStarts; j++){ // columns
            starts.read( reinterpret_cast<char*>( &startDoubles ), sizeof startDoubles );
            arrayCPU[j][i] = startDoubles;
        }
    }
    starts.close();

     // set every thread's input parameters to a set of final values from CPU calculations for use as a good starting point
    for(int i = 0; i < numThreads; i++){
        int row = mt_rand() % numStarts;

        double tripTime = arrayCPU[row][13];

        double alpha = arrayCPU[row][10];

        double beta = arrayCPU[row][11];

        double zeta = arrayCPU[row][12];

        coefficients<double> testcoeff;
        for(int j = 0; j < testcoeff.gammaSize; j++){
        testcoeff.gamma[j] = arrayCPU[row][j];
        }

        for(int j = 0; j < testcoeff.tauSize; j++){
        testcoeff.tau[j] =  arrayCPU[row][j+7];
        }

        for(int j = 0; j < testcoeff.coastSize; j++){
        testcoeff.coast[j] = arrayCPU[row][j+14];
        }

        rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

        inputParameters[i].startParams = example;
    }

    // set every thread's input parameters to random values within a reasonable range
    /*
    for(int i = 0; i < numThreads; i++){ 
        double tripTime = 365*24*3600*(std::rand() % 10001 / 10000.0 + 1.0);
        double alpha = (mt_rand() % 629) / 100.0 - 3.14;
        double beta = (mt_rand() % 629) / 100.0 - 3.14;
        double zeta = (mt_rand() % 315) / 100.0 - 1.57;

        coefficients<double> testcoeff;
        for(int j = 0; j < testcoeff.gammaSize; j++){
            testcoeff.gamma[j] = mt_rand() % 201/10.0 - 10.0;
        }
        for(int j = 0; j < testcoeff.tauSize; j++){
            testcoeff.tau[j] = mt_rand() % 201/10.0 - 10.0;
        }
        for(int j = 0; j < testcoeff.coastSize; j++){
            testcoeff.coast[j] = mt_rand() % 201/10.0 - 10.0;
        }
    
        rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

        inputParameters[i].startParams = example;
    }
    */


    Individual *survivors = new Individual[SURVIVOR_COUNT]; // stores the winners of the head-to-head competition
    int newInd = numThreads; // the whole population is new the first time through the loop

    // setup output of results
    std::ofstream individualDifference;
    individualDifference.open("individualDifference.csv");
    individualDifference << "posDiff" << "," << "velDiff" << "," << "r" << "," << "theta" << "," << "z" << "," << "vr" << "," << "vtheta" << "," << "vz" << "\n";
    
    for(int i = 0; i < generationsNum; i++){
        // initialize positions for the new individuals starting at the index of the first new one and going to the end of the array
        initializePosition(inputParameters + (numThreads - newInd), newInd);

        callRK(newInd, blockThreads, inputParameters + (numThreads - newInd), timeInitial, stepSize, absTol, calcPerS); // calculate trajectories for new individuals

        for(int k = 0; k < numThreads; k++){ // if we got bad results reset the Individual to random starting values (it may still be used for crossover) 
                                            // and set the final position to be way off so it gets replaced by a new Individual
            if(isnan(inputParameters[k].finalPos.r) || isnan(inputParameters[k].finalPos.theta) || isnan(inputParameters[k].finalPos.z) 
             || isnan(inputParameters[k].finalPos.vr) || isnan(inputParameters[k].finalPos.vtheta) || isnan(inputParameters[k].finalPos.vz)){
                
                std::cout << std::endl << std::endl << "NAN FOUND" << std::endl << std::endl;

                double tripTime = 365*24*3600*(std::rand() % 10001 / 10000.0 + 1.0);
                double alpha = (mt_rand() % 629) / 100.0 - 3.14;
                double beta = (mt_rand() % 629) / 100.0 - 3.14;
                double zeta = (mt_rand() % 315) / 100.0 - 1.57;
        
                coefficients<double> testcoeff;
                for(int j = 0; j < testcoeff.gammaSize; j++){
                    testcoeff.gamma[j] = mt_rand() % 201/10.0 - 10.0;
                }
                for(int j = 0; j < testcoeff.tauSize; j++){
                    testcoeff.tau[j] = mt_rand() % 201/10.0 - 10.0;
                }
                for(int j = 0; j < testcoeff.coastSize; j++){
                    testcoeff.coast[j] = mt_rand() % 201/10.0 - 10.0;
                }
            
                rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 
        
                inputParameters[k].startParams = example;

                inputParameters[k].posDiff = 1.0e10;
                inputParameters[k].velDiff = 0.0;
             }
        }

        std::shuffle(inputParameters, inputParameters + numThreads, mt_rand); // shuffle the Individiuals to use random members for the competition

        selectWinners(inputParameters, SURVIVOR_COUNT, survivors); // Choose which individuals are in survivors, not necessarrily only the best ones

        std::sort(inputParameters, inputParameters + numThreads, greaterInd); // put the individuals in order so we can replace the worst ones

        // finding the best variable to change in the best Individual
        // bestChange() TO BE USED HERE


        // Display a '.' to the terminal to show that a generation has been calculated
        // if it is not the 50th generation this serves to show that a generation was calculated and survivors selected
        // This also serves to visually seperate the generation display on the terminal screen
        std::cout << '.';

        // Display and print Individuals' pos and vel difference every 50 generations to terminal and .csv file
        if((i+1) % 50 == 0)
        {   
            // Display the best and worst Individuals in this generation
            std::cout << '\n';
            std::cout << "generation: " << i << std::endl;
            std::cout << "best:" << std::endl;
            std::cout << "\tposDiff: " << inputParameters[0].posDiff << std::endl;
            std::cout << "\tvelDiff: " << inputParameters[0].velDiff << std::endl;
            std::cout << "worst:" << std::endl;
            std::cout << "\tposDiff: " << inputParameters[numThreads - 1].posDiff << std::endl;
            std::cout << "\tvelDiff: " << inputParameters[numThreads - 1].velDiff << std::endl;

            // Append every Individual into a csv file to view progress over generations
            for(int j = 0; j < numThreads; j++)
            {
                individualDifference << inputParameters[j].posDiff << ","  << inputParameters[j].velDiff << ","
                << inputParameters[j].finalPos.r << "," << inputParameters[j].finalPos.theta << "," << inputParameters[j].finalPos.z << ","
                << inputParameters[j].finalPos.vr << "," << inputParameters[j].finalPos.vtheta << "," << inputParameters[j].finalPos.vz << "," << "\n";
            }
            individualDifference << "\n";
        }

        // the annnealing rate passed in is scaled between ANNEAL_MAX and ANNEAL_MIN depending on which generation this is
        double new_anneal =  ANNEAL_MAX - static_cast<double>(i) / (generationsNum - 1) * (ANNEAL_MAX - ANNEAL_MIN);

        newInd = crossover(survivors, inputParameters, SURVIVOR_COUNT, numThreads, new_anneal);
    }


    // output the best Individuals of the final generation, using writeTrajectoryToFile()
    // Files outputted allows plotting of solutions in matlab
    double *start = new double[OPTIM_VARS];
    double cost = 0;
    for(int i = 0; i < 10; i++){
        for(int j = 0; j < inputParameters[i].startParams.coeff.gammaSize; j++){
            start[GAMMA_OFFSET + j] = inputParameters[i].startParams.coeff.gamma[j];
        }
        for(int j = 0; j < inputParameters[i].startParams.coeff.tauSize; j++){
            start[TAU_OFFSET + j] = inputParameters[i].startParams.coeff.tau[j];
        }
        for(int j = 0; j < inputParameters[i].startParams.coeff.coastSize; j++){
            start[COAST_OFFSET + j] = inputParameters[i].startParams.coeff.coast[j];
        }
        start[TRIPTIME_OFFSET] = inputParameters[i].startParams.tripTime;
        start[ALPHA_OFFSET] = inputParameters[i].startParams.alpha;
        start[BETA_OFFSET] = inputParameters[i].startParams.beta;
        start[ZETA_OFFSET] = inputParameters[i].startParams.zeta;

        cost = inputParameters[i].posDiff; // just look at position difference here for now
        // could instead use a ratio between position and velocity differnce as done in comparison of Individuals
        writeTrajectoryToFile(start, cost, i + 1);
    }


    individualDifference.close();

    delete [] inputParameters;
    delete [] survivors;

    return calcPerS;
}

void callRK(const int numThreads, const int blockThreads, Individual *generation, double timeInitial, double stepSize, double absTol, double & calcPerS){
    
    hipEvent_t kernelStart, kernelEnd;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);

    Individual *devGeneration; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;

    // allocate memory for the parameters passed to the device
    hipMalloc((void**) &devGeneration, numThreads * sizeof(Individual));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));

    // copy values of parameters passed to device onto device
    hipMemcpy(devGeneration, generation, numThreads * sizeof(Individual), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);

    // GPU version of rk4Simple()
    hipEventRecord(kernelStart);
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devGeneration, devTimeInitial, devStepSize, devAbsTol, numThreads);
    hipEventRecord(kernelEnd);

    // copy the result of the kernel onto the host
    hipMemcpy(generation, devGeneration, numThreads * sizeof(Individual), hipMemcpyDeviceToHost);
    
    // free memory from device
    hipFree(devGeneration);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);

    float kernelT;
    
    hipEventSynchronize(kernelEnd);

    hipEventElapsedTime(&kernelT, kernelStart, kernelEnd);
    
    calcPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second
}


// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(Individual *individuals, double *timeInitial, double *startStepSize, double *absTolInput, int n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < n)
    {
        rkParameters<double> threadRKParameters = individuals[threadId].startParams; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;

        elements<double> k1, k2, k3, k4, k5, k6, k7; // k variables for Runge-Kutta calculation of y based off the spacecraft's final state

        thruster<double> NEXT = thruster<double>(1); // corresponds NEXT thruster to type 1 in thruster.h

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        bool coast; // to hold the result from calc_coast()

        elements<double> error; // holds output of previous value from rkCalc

        while(curTime < threadRKParameters.tripTime){

            coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.tripTime);
            curAccel = calc_accel(curPos.r, curPos.z, NEXT, massFuelSpent, stepSize, coast, static_cast<double>(WET_MASS));
            //curAccel = 0.;

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.tripTime, stepSize, curPos, threadRKParameters.coeff, curAccel, error, k1, k2, k3, k4, k5, k6, k7); 

            curTime += stepSize; // update the current time in the simulation
            
            stepSize *= calc_scalingFactor(curPos-error,error,absTol,stepSize); // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.tripTime - startTime) / 100){
                stepSize = (threadRKParameters.tripTime - startTime) / 100;
            }
            else if (stepSize < (threadRKParameters.tripTime - startTime) / 1000){
                stepSize = (threadRKParameters.tripTime - startTime) / 1000;
            }
            
            if((curTime + stepSize) > threadRKParameters.tripTime){
                stepSize = (threadRKParameters.tripTime - curTime); // shorten the last step to end exactly at time final
            }

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if (sqrt(pow(curPos.r,2)+pow(curPos.z,2)) < 0.5)
            {
                curPos.r = 1000;

                // output to this thread's index
                individuals[threadId].finalPos = curPos;
                individuals[threadId].posDiff = 1.0e10;
                individuals[threadId].velDiff =  0.0;

                return;
            }
        }

         // output to this thread's index
        individuals[threadId].finalPos = curPos;
        individuals[threadId].posDiff = sqrt(pow(R_FIN_AST - curPos.r, 2) + pow(THETA_FIN_AST - fmod(curPos.theta, 2 * M_PI), 2) + pow(Z_FIN_AST - curPos.z, 2));
        individuals[threadId].velDiff = sqrt(pow(VR_FIN_AST - curPos.vr, 2) + pow(VTHETA_FIN_AST - curPos.vtheta, 2) + pow(VZ_FIN_AST - curPos.vz, 2));


        return;
    }
    return;
}

__host__ void initializePosition(Individual *individuals, int size){
    for(int i=0; i<size ;i++){
        individuals[i].initialize();
    }
}

//testing functions
//------------------------------------------------------------------------------------------------------------------------------------------------------------
void rkCalcComparison(){
    int n = 5000;

    //parameter setup

    elements<double> *curPos = new elements<double>[n];
    elements<double> *hostCurPos = new elements<double>[n];
    for(int i = 0; i < n; i++){
        curPos[i].r = static_cast<double>(rand()%1001)/1000.0 + 0.5;
        curPos[i].theta = static_cast<double>(rand()%40000)/1000.0 - 20.0;
        curPos[i].z = static_cast<double>(rand()%200001)/10000000.0 - 0.001;
        curPos[i].vr = 0.000000018;
        curPos[i].vtheta = 0.00000021;
        curPos[i].vz = -0.000000002;

        hostCurPos[i] = curPos[i];
    }



    coefficients<double> testCoeff;
    for(int i = 0; i < testCoeff.gammaSize; i++){
        testCoeff.gamma[i] = 10;
    }
    for(int i = 0; i < testCoeff.tauSize; i++){
        testCoeff.tau[i] = 10;
    }
    for(int i = 0; i < testCoeff.coastSize; i++){
        testCoeff.coast[i] = 0.5;
    }
    testCoeff.coastThreshold = 0.05;

    double curTime = 1.0;
    double tripTime = 2.0;
    double stepSize = 4500.0;
    double accel = 5.0e-16;
    elements<double> *v = new elements<double>[n];
    elements<double> k1, k2, k3, k4, k5, k6, k7;


    double *devCurTime;
    double *devtripTime;
    double *devStepSize;
    double *devAccel;
    int *devN;
    elements<double> *devV;
    elements<double> *devCurPos;
    coefficients<double> *devTestCoeff;

    hipMalloc((void**) &devCurTime, sizeof(double));
    hipMalloc((void**) &devtripTime, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAccel, sizeof(double));
    hipMalloc((void**) &devN, sizeof(int));
    hipMalloc((void**) &devV, n * sizeof(elements<double>));
    hipMalloc((void**) &devCurPos, n * sizeof(elements<double>));
    hipMalloc((void**) &devTestCoeff, sizeof(coefficients<double>));

    hipMemcpy(devCurTime, &curTime, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devN, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devtripTime, &tripTime, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAccel, &accel, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCurPos, curPos, n * sizeof(elements<double>), hipMemcpyHostToDevice);
    hipMemcpy(devTestCoeff, &testCoeff, sizeof(coefficients<double>), hipMemcpyHostToDevice);
    std::cout << "memCpy on" << std::endl;
    rkCalcTest<<<n,1>>>(devCurTime, devtripTime, devStepSize, devTestCoeff, devAccel, devV, devCurPos, devN);
    std::cout << "kernel call" << std::endl;
    std::cout << curTime << std::endl;
    std::cout << tripTime << std::endl;
    std::cout << stepSize << std::endl;
    std::cout << accel << std::endl;
    hipMemcpy(v, devV, n * sizeof(elements<double>), hipMemcpyDeviceToHost);
    std::cout << "memCpy off #1" << std::endl;
    hipMemcpy(curPos, devCurPos, n * sizeof(elements<double>), hipMemcpyDeviceToHost); 
    std::cout << "memCpy off #2" << std::endl;

    elements<double> *hostV = new elements<double>[n];

    std::cout << curTime << std::endl;
    std::cout << tripTime << std::endl;
    std::cout << stepSize << std::endl;
    std::cout << accel << std::endl;
    //std::cout << testCoeff << std::endl;

    for(int i = 0; i < n; i++){
        std::cout << "i: " << i << std::endl;
        std::cout << hostV[i] << std::endl;
        std::cout << hostCurPos[i] << std::endl;

        rkCalc(curTime, tripTime, stepSize, hostCurPos[i], testCoeff, accel, hostV[i], k1, k2, k3, k4, k5, k6, k7);
    }

    double errorTol = 1e-10;
    for(int i = 0; i < n; i++){
        if(abs(v[i].r - hostV[i].r) > errorTol){
            std::cout << "Thread: " << i << std::endl;
            std::cout << "GPU v: " << v[i] << std::endl;
            std::cout << "CPU v: " << hostV[i] << std::endl;
            std::cout << "difference: " << v[i] - hostV[i] << std::endl;
            std::cout << "GPU curPos: " << curPos[i] << std::endl;
            std::cout << "CPU curPos: " << hostCurPos[i] << std::endl;
            std::cout << "difference: " << curPos[i] - hostCurPos[i] << std::endl;
        }
    }
    std::cout << "done checking for errors" <<std::endl;

    hipFree(devCurTime);
    hipFree(devtripTime);
    hipFree(devStepSize);
    hipFree(devAccel);
    hipFree(devN);
    hipFree(devV);
    hipFree(devCurPos);
    hipFree(devTestCoeff);

    delete [] curPos;
    delete [] hostCurPos;
    delete [] hostV;
    delete [] v;
}

__global__ void rkCalcTest(double *curTime, double *tripTime, double *stepSize, coefficients<double> *testCoeff, double *accel, elements<double> *v, elements<double> *curPos, int *n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < *n){
        elements<double> k1, k2, k3, k4, k5, k6, k7;
        rkCalc(*curTime, *tripTime, *stepSize, curPos[threadId], *testCoeff, *accel, v[threadId], k1, k2, k3, k4, k5, k6, k7);
    }
}