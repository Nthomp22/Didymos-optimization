#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES // for use of M_PI
#include "runge_kuttaCUDA.cuh"
#include "runge_kutta.h" // used for rkCalc()
#include "../Thrust_Files/acceleration.h" //used for calc_accel() and calc_coast()
#include "rkParameters.h" // the struct containing the values passed to rk4simple()
#include "../Earth_calculations/orbitalMotion.h"
#include "../CUDA_Files/geneticAlgorithm.h" // selectWinners()
#include "../CUDA_Files/ga_crossover.h" // crossover()
#include "../CUDA_Files/gaConstants.h" // SURVIVOR_COUNT
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file
#include <chrono>
#include <algorithm> // sort(), shuffle()
#include <random>


double optimize(const int numThreads, const int blockThreads){
    double calcPerS = 0;
    std::mt19937_64 mt_rand(time(0));
    //bool maxErrorMet = false;

    // reasonable example values for runge kutta algorithm
    /*---------------------------------------------------------------------------------------*/
     // input parameters for rk4Simple which are the same for each thread
    double timeInitial = 0; // the starting time of the trip is always defined as zero
    //double tripTime = 2.5*365.25*24*60*60; // number of years the trip takes
    
    double absTol = RK_TOL; // the tolerance is a constant number that is shared amongst all runs
    double stepSize = (orbitalPeriod - timeInitial) / MAX_NUMSTEPS; // the starting step size- same for each run- note that the current step size varies throughout each run
    
    //for setting every thread's parameters to the same values
  
    /*---------------------------------------------------------------------------------------*/


    Individual *inputParameters = new Individual[numThreads]; // contains all input parameters besides those which are always common amongst every thread


    for(int i = 0; i < numThreads; i++){ // set every thread's input parameters
        double tripTime = 365*24*3600*(std::rand() % 10001 / 10000.0 + 1.0);
        double alpha = (mt_rand() % 629) / 100.0 - 3.14;
        double beta = (mt_rand() % 629) / 100.0 - 3.14;
        double zeta = (mt_rand() % 315) / 100.0 - 1.57;

        coefficients<double> testcoeff;
        for(int j = 0; j < testcoeff.gammaSize; j++){
            testcoeff.gamma[j] = mt_rand() % 201/10.0 - 10.0;
        }
        for(int j = 0; j < testcoeff.tauSize; j++){
            testcoeff.tau[j] = mt_rand() % 201/10.0 - 10.0;
        }
        for(int j = 0; j < testcoeff.coastSize; j++){
            testcoeff.coast[j] = mt_rand() % 201/10.0 - 10.0;
        }
    
        //elements<double> earth = earthInitial(tripTime);
        //elements<double> spaceTest(earth.r+ESOI*cos(alpha), earth.theta+asin(sin(M_PI-alpha)*ESOI/earth.r), earth.z,
            //earth.vr+cos(zeta)*sin(beta)*vEscape, earth.vtheta+cos(zeta)*cos(beta)*vEscape, earth.vz+sin(zeta)*vEscape);
    
        rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

        inputParameters[i].startParams = example;
    }


    Individual *survivors = new Individual[SURVIVOR_COUNT];
    int newInd = numThreads; // the whole population is new the first time through the loop

    // printing individual pos and vel difference data to a csv
    std::ofstream individualDifference;
    individualDifference.open("individualDifference.csv");
    individualDifference << "posDiff" << "," << "velDiff" << "," << "r" << "," << "theta" << "," << "z" << "," << "vr" << "," << "vtheta" << "," << "vz" << "\n";

    //while(!maxErrorMet){
    for(int i = 0; i < generationsNum; i++){
        auto start = std::chrono::high_resolution_clock::now();
        initializePosition(inputParameters + (numThreads - newInd), newInd); // initialize positions for new individuals
        
        // testing
        /*
        std::cout << "generation " << i << std::endl << std::endl;
        for(int j = 0; j < numThreads; j++){ 
            std::cout << inputParameters[j].startParams.y0 << std::endl;
        }
        */
        auto RK = std::chrono::high_resolution_clock::now();
        callRK(newInd, blockThreads, inputParameters + (numThreads - newInd), timeInitial, stepSize, absTol, calcPerS); // calculate trajectories for new individuals
        
        auto NaNCheck = std::chrono::high_resolution_clock::now();
        for(int k = 0; k < numThreads; k++){ // if we got bad results reset the Individual to random starting values (it may still be used for crossover) 
                                            // and set the final position to be way off so it gets replaced by a new Individual
            if(isnan(inputParameters[k].finalPos.r) || isnan(inputParameters[k].finalPos.theta) || isnan(inputParameters[k].finalPos.z) 
             || isnan(inputParameters[k].finalPos.vr) || isnan(inputParameters[k].finalPos.vtheta) || isnan(inputParameters[k].finalPos.vz)){
                
                std::cout << std::endl << std::endl << "NAN FOUND" << std::endl << std::endl;

                double tripTime = 365*24*3600*(std::rand() % 10001 / 10000.0 + 1.0);
                double alpha = (mt_rand() % 629) / 100.0 - 3.14;
                double beta = (mt_rand() % 629) / 100.0 - 3.14;
                double zeta = (mt_rand() % 315) / 100.0 - 1.57;
        
                coefficients<double> testcoeff;
                for(int j = 0; j < testcoeff.gammaSize; j++){
                    testcoeff.gamma[j] = mt_rand() % 201/10.0 - 10.0;
                }
                for(int j = 0; j < testcoeff.tauSize; j++){
                    testcoeff.tau[j] = mt_rand() % 201/10.0 - 10.0;
                }
                for(int j = 0; j < testcoeff.coastSize; j++){
                    testcoeff.coast[j] = mt_rand() % 201/10.0 - 10.0;
                }
            
                rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 
        
                inputParameters[k].startParams = example;

                inputParameters[k].posDiff = 1.0e10;
                inputParameters[k].velDiff = 0.0;
             }
        }
        auto shuffleT = std::chrono::high_resolution_clock::now();
        std::shuffle(inputParameters, inputParameters + numThreads, mt_rand);

        auto competition = std::chrono::high_resolution_clock::now();
        selectWinners(inputParameters, SURVIVOR_COUNT, survivors);

        auto sort = std::chrono::high_resolution_clock::now();
        std::sort(inputParameters, inputParameters + numThreads, greaterInd);
        
        auto display = std::chrono::high_resolution_clock::now();
        std::cout << "generation: " << i << std::endl;
        std::cout << "best:" << std::endl;
        std::cout << "posDiff: " << inputParameters[0].posDiff << std::endl;
        std::cout << "velDiff: " << inputParameters[0].velDiff << std::endl;
        std::cout << "finalPos: " <<inputParameters[0].finalPos << std::endl;
        std::cout << "worst:" << std::endl;
        std::cout << "posDiff: " << inputParameters[numThreads - 1].posDiff << std::endl;
        std::cout << "velDiff: " << inputParameters[numThreads - 1].velDiff << std::endl;
        std::cout << "finalPos: " <<inputParameters[numThreads - 1].finalPos << std::endl << std::endl;


        // For csv file "individualDifference.csv"
        //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        if(i % 50 == 0)
        {   
            for(int j = 0; j < numThreads; j++)
            {
                individualDifference << inputParameters[j].posDiff << ","  << inputParameters[j].velDiff << "," << inputParameters[j].finalPos.r << "," <<
                 inputParameters[j].finalPos.theta << "," << inputParameters[j].finalPos.z << "," << inputParameters[j].finalPos.vr << "," << 
                 inputParameters[j].finalPos.vtheta << "," << inputParameters[j].finalPos.vz << "," << "\n";
            }
            individualDifference << "\n";
        }
        //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


        auto crossoverT = std::chrono::high_resolution_clock::now();
        newInd = crossover(survivors, inputParameters, SURVIVOR_COUNT, numThreads);
        auto end = std::chrono::high_resolution_clock::now();


        // display timing metrics
        
        std::chrono::duration<double> elapsedTime = RK - start;
        std::cout << "Execution speeds (seconds):" << std::endl;
        std::cout << "initializePosition(): " << elapsedTime.count() << std::endl;
        elapsedTime = NaNCheck - RK;
        std::cout << "callRK(): " << elapsedTime.count() << std::endl;
        elapsedTime = shuffleT - NaNCheck;
        std::cout << "NaN check: " << elapsedTime.count() << std::endl;
        elapsedTime = competition - shuffleT;
        std::cout << "shuffle(): " << elapsedTime.count() << std::endl;
        elapsedTime = sort - competition;
        std::cout << "selectWinners(): " << elapsedTime.count() << std::endl;
        elapsedTime = display - sort;
        std::cout << "sort(): " << elapsedTime.count() << std::endl;
        elapsedTime = crossoverT - display;
        std::cout << "display(): " << elapsedTime.count() << std::endl;
        elapsedTime = end - crossoverT;
        std::cout << "crossover(): " << elapsedTime.count() << std::endl << std::endl;
        
    }

    individualDifference.close();

    delete [] inputParameters;
    delete [] survivors;

    return calcPerS;
}

void callRK(const int numThreads, const int blockThreads, Individual *generation, double timeInitial, double stepSize, double absTol, double & calcPerS){
    

    auto start2 = std::chrono::high_resolution_clock::now();
    //events for timing functions
    hipEvent_t Malloc_e, MemCpyDev_e, Kernel_e, MemCpyHost_e, MemCpyHostStop_e;
    hipEventCreate(&Malloc_e);
    hipEventCreate(&MemCpyDev_e);
    hipEventCreate(&Kernel_e);
    hipEventCreate(&MemCpyHost_e);
    hipEventCreate(&MemCpyHostStop_e);

    auto indiv = std::chrono::high_resolution_clock::now();
    Individual *devGeneration; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;

    auto allocating = std::chrono::high_resolution_clock::now();
    // allocate memory for the parameters passed to the device
    hipEventRecord(Malloc_e);
    hipMalloc((void**) &devGeneration, numThreads * sizeof(Individual));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));

    auto copyParam = std::chrono::high_resolution_clock::now();
    // copy values of parameters passed to device onto device
    hipEventRecord(MemCpyDev_e);
    hipMemcpy(devGeneration, generation, numThreads * sizeof(Individual), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);


    auto rkSIM = std::chrono::high_resolution_clock::now();
    // GPU version of rk4Simple()
    hipEventRecord(Kernel_e);
    //std::cout << "Starting kernel with: <<<" << (numThreads+blockThreads-1)/blockThreads << "," << blockThreads << ">>>\n";
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devGeneration, devTimeInitial, devStepSize, devAbsTol, numThreads);


    auto copyRes = std::chrono::high_resolution_clock::now();
    // copy the result of the kernel onto the host
    hipEventRecord(MemCpyHost_e);
    hipMemcpy(generation, devGeneration, numThreads * sizeof(Individual), hipMemcpyDeviceToHost);
    hipEventRecord(MemCpyHostStop_e);
    

    auto freeMem = std::chrono::high_resolution_clock::now();
    // free memory from device
    hipFree(devGeneration);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);
    hipFree(devGeneration);



    auto rkSIM_CPU = std::chrono::high_resolution_clock::now();
    // CPU version of rk4Simple()
    // only calculate once since all input parameters are currently the same
    //elements<double> rk4SimpleOutput;
    //inputParameters[0].parametersRK4Simple(timeInitial, stepSize, absTol, rk4SimpleOutput);

    /*elements<double> *rk4SimpleOutput = new elements<double>[numThreads];
    
    for(int i = 0; i < numThreads; i++){
        generation[i].startParams.parametersRK4Simple(timeInitial, stepSize, absTol, rk4SimpleOutput[i]);
          //std::cout << rk4SimpleOutput[i];
    }
    */
    /*
    std::cout << "CPU Calculation of " << numThreads << " RK Calculations took: " << std::chrono::duration_cast<std::chrono::milliseconds>(elapsed_time).count() << " ms" << std::endl;
    std::cout << "CPU Calculations: " << numThreads / (std::chrono::duration_cast<std::chrono::milliseconds>(elapsed_time).count()/1000.0) << " RK Calcs / second" <<  std::endl;
    */

   /*
    auto errorFinding = std::chrono::high_resolution_clock::now();
    // compare every GPU result with the one CPU result
    double maxError = 1e-12; // how much difference is allowable between the CPU and GPU results
    bool errorFound = false;
    for(int i = 0; i < numThreads; i++){
        if(!generation[i].finalPos.compare(rk4SimpleOutput[i],maxError)){
            errorFound = true;
        }

        if(errorFound){
            std::cout << "!!ERROR FOUND!!" << std::endl;
            std::cout << "CPU output " << i << std::endl;
            std::cout << rk4SimpleOutput[i] << std::endl;
            std::cout << "GPU output " << i << std::endl;
            std::cout << generation[i].finalPos << std::endl;
            std::cout << "Diff: " << std::endl;
            std::cout << generation[i].finalPos-rk4SimpleOutput[i] << std::endl;

            errorFound = false;
        }

        //testing
     
        std::cout << "final position" << generation[i].finalPos << std::endl;
        std::cout << "position difference" << generation[i].posDiff << std::endl;
        std::cout << "velocity difference" << generation[i].velDiff << std::endl << std::endl << std::endl;
   
    }
     */

    
    auto resultCheck = std::chrono::high_resolution_clock::now();
    float mallocT, memCpyDevT, kernelT, memCpyHostT;
    
    hipEventSynchronize(MemCpyHostStop_e);

    hipEventElapsedTime(&mallocT, Malloc_e, MemCpyDev_e);
    hipEventElapsedTime(&memCpyDevT, MemCpyDev_e, Kernel_e);
    hipEventElapsedTime(&kernelT, Kernel_e, MemCpyHost_e);
    hipEventElapsedTime(&memCpyHostT, MemCpyHost_e, MemCpyHostStop_e);
    
    double rkPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second

    /*
    std::cout << "Device memory allocation time: " << mallocT << " ms" << std::endl;
    std::cout << "Device memory copy time: " << memCpyDevT << " ms" << std::endl;
    std::cout << "Host memory copy time: " << memCpyHostT << " ms" << std::endl;
    std::cout << "Kernel time: " << kernelT << " ms" << std::endl;
    std::cout << "Runge Kutta calculations per second: " << rkPerS << " /s" << std::endl;
    */

    calcPerS = rkPerS;
    auto end = std::chrono::high_resolution_clock::now();

    
    //delete [] rk4SimpleOutput;


    // display timing metrics

    std::chrono::duration<double> elapsedTime = indiv - start2;
    std::cout << "Execution speeds (seconds):" << std::endl;
    std::cout << "start: " << elapsedTime.count() << std::endl;

    elapsedTime = allocating - indiv;
    std::cout << "individuals: " << elapsedTime.count() << std::endl;

    elapsedTime = copyParam - allocating;
    std::cout << "allocating: " << elapsedTime.count() << std::endl;

    elapsedTime = rkSIM - copyParam;
    std::cout << "Copy parameters: " << elapsedTime.count() << std::endl;

    elapsedTime = copyRes - rkSIM;
    std::cout << "RK simple: " << elapsedTime.count() << std::endl;

    elapsedTime = freeMem - copyRes;
    std::cout << "copy results: " << elapsedTime.count() << std::endl;

    elapsedTime = rkSIM_CPU - freeMem;
    std::cout << "Freeing memory: " << elapsedTime.count() << std::endl;

    elapsedTime = resultCheck - rkSIM_CPU;
    std::cout << "RK simple CPU: " << elapsedTime.count() << std::endl;

    elapsedTime = end - resultCheck;
    std::cout << "result check: " << elapsedTime.count() << std::endl << std::endl;

}



// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(Individual *individuals, double *timeInitial, double *startStepSize, double *absTolInput, int n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < n)
    {
        rkParameters<double> threadRKParameters = individuals[threadId].startParams; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;


        elements<double> k1, k2, k3, k4, k5, k6, k7; // k variables for Runge-Kutta calculation of y based off the spacecraft's final state

        thruster<double> NEXT = thruster<double>(1); // corresponds NEXT thruster to type 1 in thruster.h

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        //double deltaT; // change in time for calc_accel()

        bool coast; // to hold the result from calc_coast()

        elements<double> error; // holds output of previous value from rkCalc

        while(curTime < threadRKParameters.tripTime){
            //deltaT = stepSize;

            coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.tripTime);
            curAccel = calc_accel(curPos.r, curPos.z, NEXT, massFuelSpent, stepSize, coast, static_cast<double>(WET_MASS));
            //curAccel = 0.;

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.tripTime, stepSize, curPos, threadRKParameters.coeff, curAccel, error, k1, k2, k3, k4, k5, k6, k7); 

            curTime += stepSize; // update the current time in the simulation
            
            stepSize *= calc_scalingFactor(curPos-error,error,absTol,stepSize)/2; // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.tripTime - startTime) / 1000){
                stepSize = (threadRKParameters.tripTime - startTime) / 1000;
            }
            else if (stepSize < ((threadRKParameters.tripTime - startTime) / 10000)){
                stepSize = (threadRKParameters.tripTime - startTime) / 10000;
            }
            
            if((curTime + stepSize) > threadRKParameters.tripTime){
                stepSize = (threadRKParameters.tripTime - curTime); // shorten the last step to end exactly at time final
            }

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if (curPos.r < 0.5)
            {
                curPos.r = 1000;
            }
        }
        individuals[threadId].finalPos = curPos; // output to this thread's index

        individuals[threadId].posDiff =  sqrt(pow(R_FIN_AST - curPos.r, 2) + pow(THETA_FIN_AST - fmod(curPos.theta, 2 * M_PI), 2) + pow(Z_FIN_AST - curPos.z, 2));
        individuals[threadId].velDiff =  sqrt(pow(VR_FIN_AST - curPos.vr, 2) + pow(VTHETA_FIN_AST - curPos.vtheta, 2) + pow(VZ_FIN_AST - curPos.vz, 2));
        return;
    }
    return;
}



//testing functions
void rkCalcComparison(){
    int n = 5000;

    //parameter setup

    elements<double> *curPos = new elements<double>[n];
    elements<double> *hostCurPos = new elements<double>[n];
    for(int i = 0; i < n; i++){
        curPos[i].r = static_cast<double>(rand()%1001)/1000.0 + 0.5;
        curPos[i].theta = static_cast<double>(rand()%40000)/1000.0 - 20.0;
        curPos[i].z = static_cast<double>(rand()%200001)/10000000.0 - 0.001;
        curPos[i].vr = 0.000000018;
        curPos[i].vtheta = 0.00000021;
        curPos[i].vz = -0.000000002;

        hostCurPos[i] = curPos[i];
    }



    coefficients<double> testCoeff;
    for(int i = 0; i < testCoeff.gammaSize; i++){
        testCoeff.gamma[i] = 10;
    }
    for(int i = 0; i < testCoeff.tauSize; i++){
        testCoeff.tau[i] = 10;
    }
    for(int i = 0; i < testCoeff.coastSize; i++){
        testCoeff.coast[i] = 0.5;
    }
    testCoeff.coastThreshold = 0.05;

    double curTime = 1.0;
    double tripTime = 2.0;
    double stepSize = 4500.0;
    double accel = 5.0e-16;
    elements<double> *v = new elements<double>[n];
    elements<double> k1, k2, k3, k4, k5, k6, k7;


    double *devCurTime;
    double *devtripTime;
    double *devStepSize;
    double *devAccel;
    int *devN;
    elements<double> *devV;
    elements<double> *devCurPos;
    coefficients<double> *devTestCoeff;

    hipMalloc((void**) &devCurTime, sizeof(double));
    hipMalloc((void**) &devtripTime, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAccel, sizeof(double));
    hipMalloc((void**) &devN, sizeof(int));
    hipMalloc((void**) &devV, n * sizeof(elements<double>));
    hipMalloc((void**) &devCurPos, n * sizeof(elements<double>));
    hipMalloc((void**) &devTestCoeff, sizeof(coefficients<double>));

    hipMemcpy(devCurTime, &curTime, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devN, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devtripTime, &tripTime, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAccel, &accel, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCurPos, curPos, n * sizeof(elements<double>), hipMemcpyHostToDevice);
    hipMemcpy(devTestCoeff, &testCoeff, sizeof(coefficients<double>), hipMemcpyHostToDevice);
    std::cout << "memCpy on" << std::endl;
    rkCalcTest<<<n,1>>>(devCurTime, devtripTime, devStepSize, devTestCoeff, devAccel, devV, devCurPos, devN);
    std::cout << "kernel call" << std::endl;
    std::cout << curTime << std::endl;
    std::cout << tripTime << std::endl;
    std::cout << stepSize << std::endl;
    std::cout << accel << std::endl;
    hipMemcpy(v, devV, n * sizeof(elements<double>), hipMemcpyDeviceToHost);
    std::cout << "memCpy off #1" << std::endl;
    hipMemcpy(curPos, devCurPos, n * sizeof(elements<double>), hipMemcpyDeviceToHost); 
    std::cout << "memCpy off #2" << std::endl;

    elements<double> *hostV = new elements<double>[n];

    std::cout << curTime << std::endl;
    std::cout << tripTime << std::endl;
    std::cout << stepSize << std::endl;
    std::cout << accel << std::endl;
    //std::cout << testCoeff << std::endl;

    for(int i = 0; i < n; i++){
        std::cout << "i: " << i << std::endl;
        std::cout << hostV[i] << std::endl;
        std::cout << hostCurPos[i] << std::endl;

        rkCalc(curTime, tripTime, stepSize, hostCurPos[i], testCoeff, accel, hostV[i], k1, k2, k3, k4, k5, k6, k7);
    }

    double errorTol = 1e-10;
    for(int i = 0; i < n; i++){
        if(abs(v[i].r - hostV[i].r) > errorTol){
            std::cout << "Thread: " << i << std::endl;
            std::cout << "GPU v: " << v[i] << std::endl;
            std::cout << "CPU v: " << hostV[i] << std::endl;
            std::cout << "difference: " << v[i] - hostV[i] << std::endl;
            std::cout << "GPU curPos: " << curPos[i] << std::endl;
            std::cout << "CPU curPos: " << hostCurPos[i] << std::endl;
            std::cout << "difference: " << curPos[i] - hostCurPos[i] << std::endl;
        }
    }
    std::cout << "done checking for errors" <<std::endl;

    hipFree(devCurTime);
    hipFree(devtripTime);
    hipFree(devStepSize);
    hipFree(devAccel);
    hipFree(devN);
    hipFree(devV);
    hipFree(devCurPos);
    hipFree(devTestCoeff);

    delete [] curPos;
    delete [] hostCurPos;
    delete [] hostV;
    delete [] v;
}

__global__ void rkCalcTest(double *curTime, double *tripTime, double *stepSize, coefficients<double> *testCoeff, double *accel, elements<double> *v, elements<double> *curPos, int *n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < *n){
        elements<double> k1, k2, k3, k4, k5, k6, k7;
        rkCalc(*curTime, *tripTime, *stepSize, curPos[threadId], *testCoeff, *accel, v[threadId], k1, k2, k3, k4, k5, k6, k7);
    }
}

__host__ void initializePosition(Individual *individuals, int size){
    for(int i=0; i<size ;i++){
        individuals[i].initialize();
    }
}