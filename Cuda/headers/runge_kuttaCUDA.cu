#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES // for use of M_PI
#include "runge_kuttaCUDA.cuh"
#include "runge_kutta.h" // used for rkCalc()
#include "acceleration.h" //used for calc_accel() and calc_coast()
#include "rkParameters.h" // the struct containing the values passed to rk4simple()
#include "orbitalMotion.h"
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file

double callRK(const int numThreads, const int blockThreads){

    // input parameters for rk4Simple which are the same for each thread
    double timeInitial = 0; // the starting time of the trip is always defined as zero
    double absTol = RK_TOL; // the tolerance is a constant number that is shared amongst all runs
    double stepSize = (orbitalPeriod - timeInitial) / MAX_NUMSTEPS; // the starting step size- same for each run- note that the current step size varies throughout each run

    elements<double> *finalPos = new elements<double>[numThreads]; // to store the output of final position and velocity for each run

    // reasonable example values for runge kutta algorithm
    /*-------------------------------------------------------------------------------------*/
    //double timeFinal = 2.5; // number of years the trip takes
    //double timeFinal = 2.5*365.25*24*60*60; // number of years the trip takes
    double timeFinal = 75178800-3600;
    
    
    //for setting every thread's parameters to the same values
    /*
    double gamma[] = {10, 10, 10, 10, 10, 10, 10, 10, 10};
    double tau[] = {3, 3, 3, 3, 3};
    double coast[] = {5, 5, 5, 5, 5};

    elements<double> earth = earthInitial(timeFinal);

    // timeFinal, accel, wetMass, 
    // r, theta, z, 
    // vr, vtheta, vz, 
    // gamma[], tau[], coast[], coastThreshold0
    rkParameters<double> example // contains all input parameters besides those which are always common amongst every thread
    (timeFinal, 0.0, WET_MASS, 
    earth.r+ESOI*cos(10), earth.theta+asin(sin(M_PI-10)*ESOI/earth.r), earth.z,
    earth.vr+sin(3)*vEscape, earth.vtheta+cos(3)*vEscape, earth.vz,
    gamma, tau, coast, 0.05);
    */

    rkParameters<double> *inputParameters = new rkParameters<double>[numThreads]; // contains all input parameters besides those which are always common amongst every thread

    for(int i = 0; i < numThreads; i++){ // set every thread's input parameters
        
        /*double gamma[] = {i, i, i, i, i, i, i, i, i};
        double tau[] = {i%4, i%4, i%4, i%4, i%4};
        double coast[] = {i%6, i%6, i%6, i%6, i%6};    
    
        elements<double> earth = earthInitial(timeFinal);
        
        inputParameters[i] = rkParameters<double>(timeFinal - i / 32, 0.0, WET_MASS,
        earth.r+ESOI*cos(i), earth.theta+asin(sin(M_PI-i)*ESOI/earth.r), earth.z,
        earth.vr+sin(i%4)*vEscape, earth.vtheta+cos(i%4)*vEscape, earth.vz,
        gamma, tau, coast, 0.005 * i);*/

        double gamma[] = {10, 10, 10, 10, 10, 10, 10, 10, 10};
        double tau[] = {5, 5, 5, 5, 5};
        double coast[] = {3, 3, 3, 3, 3};    
    
        elements<double> earth = earthInitial(timeFinal);
        
        inputParameters[i] = rkParameters<double>(timeFinal, WET_MASS,
        earth.r+ESOI*cos(0.5), earth.theta+asin(sin(M_PI-0.5)*ESOI/earth.r), earth.z,
        earth.vr+sin(0.1)*vEscape, earth.vtheta+cos(0.1)*vEscape, earth.vz,
        gamma, tau, coast, 0.05);
        
        // doesn't work
        //inputParameters[i] = example;
    }
    /*-------------------------------------------------------------------------------------*/

    
    hipEvent_t Malloc_e, MemCpyDev_e, Kernel_e, MemCpyHost_e, MemCpyHostStop_e;
    hipEventCreate(&Malloc_e);
    hipEventCreate(&MemCpyDev_e);
    hipEventCreate(&Kernel_e);
    hipEventCreate(&MemCpyHost_e);
    hipEventCreate(&MemCpyHostStop_e);
    

    rkParameters<double> *devInputParameters; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;
    elements<double> *devFinalPos;
   
    // allocate memory for the parameters passed to the device
    hipEventRecord(Malloc_e);
    hipMalloc((void**) &devInputParameters, numThreads * sizeof(rkParameters<double>));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));
    hipMalloc((void**) &devFinalPos, numThreads * sizeof(elements<double>));

    // copy values of parameters passed to device onto device
    hipEventRecord(MemCpyDev_e);
    hipMemcpy(devInputParameters, inputParameters, numThreads * sizeof(rkParameters<double>), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);

    // GPU version of rk4Simple()
    hipEventRecord(Kernel_e);
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devInputParameters, devTimeInitial, devStepSize, devAbsTol, devFinalPos, numThreads);


    // copy the result of the kernel onto the host
    hipEventRecord(MemCpyHost_e);
    hipMemcpy(finalPos, devFinalPos, numThreads * sizeof(elements<double>), hipMemcpyDeviceToHost);
    hipEventRecord(MemCpyHostStop_e);
    
    // free memory from device
    hipFree(devInputParameters);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);

    // CPU version of rk4Simple()
    elements<double> *rk4SimpleOutput = new elements<double>[numThreads];
    for(int i = 0; i < numThreads; i++){
        inputParameters[i].parametersRK4Simple(timeInitial, stepSize, absTol, rk4SimpleOutput[i]);
    }

    // display final r, theta, z, vr, vtheta, and vz
    double maxError = 0.001; // how much difference is allowable between the CPU and GPU results
    bool errorFound = false;
    for(int i = 0; i < numThreads; i++){
        if(abs(rk4SimpleOutput[i].r - finalPos[i].r) > maxError){
            errorFound = true;
        }
        else if(abs(rk4SimpleOutput[i].theta - finalPos[i].theta) > maxError){
            errorFound = true;
        }
        else if(abs(rk4SimpleOutput[i].z - finalPos[i].z) > maxError){
            errorFound = true;
        }
        else if(abs(rk4SimpleOutput[i].vr - finalPos[i].vr) > maxError){
            errorFound = true;
        }
        else if(abs(rk4SimpleOutput[i].vtheta - finalPos[i].vtheta) > maxError){
            errorFound = true;
        }
        else if(abs(rk4SimpleOutput[i].vz - finalPos[i].vz) > maxError){
            errorFound = true;
        }

        if(errorFound){
            std::cout << "!!ERROR FOUND!!" << std::endl;
            std::cout << "CPU output " << i << std::endl;
            std::cout << rk4SimpleOutput[i] << std::endl;
            std::cout << "GPU output " << i << std::endl;
            std::cout << finalPos[i] << std::endl;

            errorFound = false;
        }
    }

    float mallocT, memCpyDevT, kernelT, memCpyHostT;
    
    hipEventSynchronize(MemCpyHostStop_e);

    hipEventElapsedTime(&mallocT, Malloc_e, MemCpyDev_e);
    hipEventElapsedTime(&memCpyDevT, MemCpyDev_e, Kernel_e);
    hipEventElapsedTime(&kernelT, Kernel_e, MemCpyHost_e);
    hipEventElapsedTime(&memCpyHostT, MemCpyHost_e, MemCpyHostStop_e);
    
    double rkPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second

    std::cout << "Device memory allocation time: " << mallocT << " ms" << std::endl;
    std::cout << "Device memory copy time: " << memCpyDevT << " ms" << std::endl;
    std::cout << "Host memory copy time: " << memCpyHostT << " ms" << std::endl;
    std::cout << "Kernel time: " << kernelT << " ms" << std::endl;
    std::cout << "Runge Kutta calculations per second: " << rkPerS << " /s" << std::endl;

    delete [] rk4SimpleOutput;
    delete [] finalPos;
    delete [] inputParameters;
    
    return rkPerS;
}

// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(rkParameters<double> * rkParametersList, double *timeInitial, double *startStepSize, double *absTolInput, elements<double> *finalPos, int n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId <= n)
    {
        rkParameters<double> threadRKParameters = rkParametersList[threadId]; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;


        elements<double> k1, k2, k3, k4, k5, k6, k7; // k variables for Runge-Kutta calculation of y based off the spacecraft's final state

        thruster<double> NEXT = thruster<double>(1); // corresponds NEXT thruster to type 1 in thruster.h

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        //double deltaT; // change in time for calc_accel()

        double coast; // to hold the result from calc_coast()

        elements<double> v; // holds output of previous value from rkCalc

        while(curTime < threadRKParameters.timeFinal){
            //deltaT = stepSize;

            coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.timeFinal);
            curAccel = calc_accel(curPos.r, curPos.z, NEXT, massFuelSpent, stepSize, coast, threadRKParameters.wetMass);

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.timeFinal, stepSize, curPos, threadRKParameters.coeff, curAccel, v, curPos); 

            curTime += stepSize; // update the current time in the simulation
            stepSize *= calc_scalingFactor(v,curPos-v,absTol,stepSize); // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.timeFinal - startTime) / 100){
                stepSize = (threadRKParameters.timeFinal - startTime) / 100;
            }
            else if (stepSize < ((threadRKParameters.timeFinal - startTime) / 10000)){
                stepSize = (threadRKParameters.timeFinal - startTime) / 10000;
            }

            if((curTime + stepSize) > threadRKParameters.timeFinal)
                stepSize = (threadRKParameters.timeFinal - curTime); // shorten the last step to end exactly at time final

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if (curPos.r < 0.5)
            {
                curPos.r = 1000;
            }
        }

    finalPos[threadId] = curPos; // output to this thread's index
    }
}