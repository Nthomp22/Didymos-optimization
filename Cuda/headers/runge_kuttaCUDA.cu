#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES // for use of M_PI
#include "runge_kuttaCUDA.cuh"
#include "runge_kutta.h" // used for rkCalc()
#include "acceleration.h" //used for calc_accel() and calc_coast()
#include "rkParameters.h" // the struct containing the values passed to rk4simple()
#include "orbitalMotion.h"
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file
#include <chrono>

double optimize(const int numThreads, const int blockThreads){
    double calcPerS = 0;

    bool maxErrorMet = false;
    elements<double> *output;

    // reasonable example values for runge kutta algorithm
    /*---------------------------------------------------------------------------------------*/
     // input parameters for rk4Simple which are the same for each thread
    double timeInitial = 0; // the starting time of the trip is always defined as zero
    //double timeFinal = 2.5*365.25*24*60*60; // number of years the trip takes
    double timeFinal = 75178800-3600;
    double absTol = RK_TOL; // the tolerance is a constant number that is shared amongst all runs
    double stepSize = (orbitalPeriod - timeInitial) / MAX_NUMSTEPS; // the starting step size- same for each run- note that the current step size varies throughout each run
    
    //for setting every thread's parameters to the same values
    coefficients<double> testcoeff;
    for(int i = 0; i < testcoeff.gammaSize; i++){
        testcoeff.gamma[i] = 10;
    }
    for(int i = 0; i < testcoeff.tauSize; i++){
        testcoeff.tau[i] = 10;
    }
    for(int i = 0; i < testcoeff.coastSize; i++){
        testcoeff.coast[i] = 10;
    }
    testcoeff.coastThreshold = .5;

    elements<double> earth = earthInitial(timeFinal);
    elements<double> spaceTest(earth.r+ESOI*cos(0.5), earth.theta+asin(sin(M_PI-0.5)*ESOI/earth.r), earth.z,
                               earth.vr+sin(0.75)*vEscape, earth.vtheta+cos(0.75)*vEscape, earth.vz);

    rkParameters<double> example(timeFinal, WET_MASS,spaceTest, testcoeff); 
    /*---------------------------------------------------------------------------------------*/


    rkParameters<double> *inputParameters = new rkParameters<double>[numThreads]; // contains all input parameters besides those which are always common amongst every thread


    for(int i = 0; i < numThreads; i++){ // set every thread's input parameters
        
        /*double gamma[] = {i, i, i, i, i, i, i, i, i};
        double tau[] = {i%4, i%4, i%4, i%4, i%4};
        double coast[] = {i%6, i%6, i%6, i%6, i%6};    
    
        elements<double> earth = earthInitial(timeFinal);
        
        inputParameters[i] = rkParameters<double>(timeFinal - i / 32, 0.0, WET_MASS,
        earth.r+ESOI*cos(i), earth.theta+asin(sin(M_PI-i)*ESOI/earth.r), earth.z,
        earth.vr+sin(i%4)*vEscape, earth.vtheta+cos(i%4)*vEscape, earth.vz,
        gamma, tau, coast, 0.005 * i);*/

        /*double gamma[] = {10, 10, 10, 10, 10, 10, 10, 10, 10};
        double tau[] = {5, 5, 5, 5, 5};
        double coast[] = {3, 3, 3, 3, 3};    
    
        elements<double> earth = earthInitial(timeFinal);
        
        inputParameters[i] = rkParameters<double>(timeFinal, WET_MASS,
        earth.r+ESOI*cos(0.5), earth.theta+asin(sin(M_PI-0.5)*ESOI/earth.r), earth.z,
        earth.vr+sin(0.1)*vEscape, earth.vtheta+cos(0.1)*vEscape, earth.vz,
        gamma, tau, coast, 0.05);*/
        
        //set all inputs to the same values
        inputParameters[i] = example;
    }

    //Check to see if the input data is all the same
    for(int i = 0; i < numThreads-1; i++){
        if(!inputParameters[i].compare(inputParameters[i+1],1.0)){
            std::cout << "Things are off in the starting set" << std::endl;
        }
    }

    //while(!maxErrorMet){
    for(int i = 0; i < 1; i++){
        output = callRK(numThreads, blockThreads, inputParameters, timeInitial, stepSize, absTol, calcPerS);
        inputParameters = getNewStarts(inputParameters, output);
        delete [] output;
    }
    delete [] inputParameters;

    return calcPerS;
}

rkParameters<double>* getNewStarts(rkParameters<double> *startParameters, elements<double> *finalPositions){
    //implement genetic algorithm
    //rkParameters<double> *newParameters = new rkParameters<double>[];

    
    //delete [] startParameters;

    return startParameters;
}

elements<double>* callRK(const int numThreads, const int blockThreads, rkParameters<double> *inputParameters, double timeInitial, double stepSize, double absTol, double & calcPerS){

    elements<double> *finalPos = new elements<double>[numThreads]; // to store the output of final position and velocity for each run
    double *pDiff= new double[numThreads]; //difference in position between spacecraft and asteroid at end of trajectory
    double *vDiff = new double[numThreads]; //difference in velocity between spacecraft and asteroid at end of trajectory
    
    //events for timing functions
    hipEvent_t Malloc_e, MemCpyDev_e, Kernel_e, MemCpyHost_e, MemCpyHostStop_e;
    hipEventCreate(&Malloc_e);
    hipEventCreate(&MemCpyDev_e);
    hipEventCreate(&Kernel_e);
    hipEventCreate(&MemCpyHost_e);
    hipEventCreate(&MemCpyHostStop_e);
    
    rkParameters<double> *devInputParameters; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;
    elements<double> *devFinalPos;
    double *devPDiff;
    double *devVDiff;


    // allocate memory for the parameters passed to the device
    hipEventRecord(Malloc_e);
    hipMalloc((void**) &devInputParameters, numThreads * sizeof(rkParameters<double>));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));
    hipMalloc((void**) &devFinalPos, numThreads * sizeof(elements<double>));
    hipMalloc((void**) &devPDiff, numThreads * sizeof(double));
    hipMalloc((void**) &devVDiff, numThreads * sizeof(double));

    // copy values of parameters passed to device onto device
    hipEventRecord(MemCpyDev_e);
    hipMemcpy(devInputParameters, inputParameters, numThreads * sizeof(rkParameters<double>), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);

    // GPU version of rk4Simple()
    hipEventRecord(Kernel_e);
    std::cout << "Starting kernel with: <<<" << (numThreads+blockThreads-1)/blockThreads << "," << blockThreads << ">>>\n";
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devInputParameters, devTimeInitial, devStepSize, devAbsTol, devFinalPos, devPDiff, devVDiff, numThreads);

    // copy the result of the kernel onto the host
    hipEventRecord(MemCpyHost_e);
    hipMemcpy(finalPos, devFinalPos, numThreads * sizeof(elements<double>), hipMemcpyDeviceToHost);
    hipMemcpy(pDiff, devPDiff, numThreads * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(vDiff, devVDiff, numThreads * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(MemCpyHostStop_e);
    
    // free memory from device
    hipFree(devInputParameters);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);
    hipFree(devPDiff);
    hipFree(devVDiff);

    // CPU version of rk4Simple()
    // only calculate once since all input parameters are currently the same
    //elements<double> rk4SimpleOutput;
    //inputParameters[0].parametersRK4Simple(timeInitial, stepSize, absTol, rk4SimpleOutput);

    elements<double> *rk4SimpleOutput = new elements<double>[numThreads];

    auto start_timer = std::chrono::high_resolution_clock::now();

    for(int i = 0; i < numThreads; i++){
        inputParameters[i].parametersRK4Simple(timeInitial, stepSize, absTol, rk4SimpleOutput[i]);
          //std::cout << rk4SimpleOutput[i];
    }

    auto elapsed_time =  std::chrono::high_resolution_clock::now() - start_timer;
    std::cout << "CPU Calculation of " << numThreads << " RK Calculations took: " << std::chrono::duration_cast<std::chrono::milliseconds>(elapsed_time).count() << " ms" << std::endl;
    std::cout << "CPU Calculations: " << numThreads / (std::chrono::duration_cast<std::chrono::milliseconds>(elapsed_time).count()/1000.0) << " RK Calcs / second" <<  std::endl;

    // compare every GPU result with the one CPU result
    double maxError = 0.01; // how much difference is allowable between the CPU and GPU results
    bool errorFound = false;
    for(int i = 0; i < numThreads; i++){
        if(!finalPos[i].compare(rk4SimpleOutput[i],maxError)){
            errorFound = true;
        }

        if(errorFound){
            std::cout << "!!ERROR FOUND!!" << std::endl;
            std::cout << "CPU output " << i << std::endl;
            std::cout << rk4SimpleOutput[i] << std::endl;
            std::cout << "GPU output " << i << std::endl;
            std::cout << finalPos[i] << std::endl;
            std::cout << "Diff: " << std::endl;
            std::cout << finalPos[i]-rk4SimpleOutput[i] << std::endl;

            errorFound = false;
        }

        //testing
        std::cout << "final position" << finalPos[i] << std::endl;
        std::cout << "position difference" << pDiff[i] << std::endl;
        std::cout << "velocity difference" << vDiff[i] << std::endl << std::endl << std::endl;
    }

    float mallocT, memCpyDevT, kernelT, memCpyHostT;
    
    hipEventSynchronize(MemCpyHostStop_e);

    hipEventElapsedTime(&mallocT, Malloc_e, MemCpyDev_e);
    hipEventElapsedTime(&memCpyDevT, MemCpyDev_e, Kernel_e);
    hipEventElapsedTime(&kernelT, Kernel_e, MemCpyHost_e);
    hipEventElapsedTime(&memCpyHostT, MemCpyHost_e, MemCpyHostStop_e);
    
    double rkPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second

    std::cout << "Device memory allocation time: " << mallocT << " ms" << std::endl;
    std::cout << "Device memory copy time: " << memCpyDevT << " ms" << std::endl;
    std::cout << "Host memory copy time: " << memCpyHostT << " ms" << std::endl;
    std::cout << "Kernel time: " << kernelT << " ms" << std::endl;
    std::cout << "Runge Kutta calculations per second: " << rkPerS << " /s" << std::endl;

    calcPerS = rkPerS;
    

    //delete [] rk4SimpleOutput;
    delete [] pDiff;
    delete [] vDiff;
    
    //return rkPerS;
    return finalPos; //make sure to delete outside of this function
}

// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(rkParameters<double> * rkParametersList, double *timeInitial, double *startStepSize, double *absTolInput, elements<double> *finalPos, double *finalPDiff, double *finalVDiff, int n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < n)
    {
        rkParameters<double> threadRKParameters = rkParametersList[threadId]; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;


        elements<double> k1, k2, k3, k4, k5, k6, k7; // k variables for Runge-Kutta calculation of y based off the spacecraft's final state

        thruster<double> NEXT = thruster<double>(1); // corresponds NEXT thruster to type 1 in thruster.h

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        //double deltaT; // change in time for calc_accel()

        double coast; // to hold the result from calc_coast()

        elements<double> v; // holds output of previous value from rkCalc

        while(curTime < threadRKParameters.timeFinal){
            //deltaT = stepSize;

            coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.timeFinal);
            curAccel = calc_accel(curPos.r, curPos.z, NEXT, massFuelSpent, stepSize, coast, threadRKParameters.wetMass);

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.timeFinal, stepSize, curPos, threadRKParameters.coeff, curAccel, v, k1, k2, k3, k4, k5, k6, k7); 

            curTime += stepSize; // update the current time in the simulation
            stepSize *= calc_scalingFactor(curPos-v,v,absTol,stepSize); // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.timeFinal - startTime) / 100){
                stepSize = (threadRKParameters.timeFinal - startTime) / 100;
            }
            else if (stepSize < ((threadRKParameters.timeFinal - startTime) / 10000)){
                stepSize = (threadRKParameters.timeFinal - startTime) / 10000;
            }

            if((curTime + stepSize) > threadRKParameters.timeFinal){
                stepSize = (threadRKParameters.timeFinal - curTime); // shorten the last step to end exactly at time final
            }

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if (curPos.r < 0.5)
            {
                curPos.r = 1000;
            }
        }
        finalPos[threadId] = curPos; // output to this thread's index
        finalPDiff[threadId] =  sqrt(pow(R_FIN_AST - curPos.r, 2) + pow(THETA_FIN_AST - curPos.theta, 2) + pow(Z_FIN_AST - curPos.z, 2));
        finalVDiff[threadId] =  sqrt(pow(VR_FIN_AST - curPos.vr, 2) + pow(VTHETA_FIN_AST - curPos.vtheta, 2) + pow(VZ_FIN_AST - curPos.vz, 2));
        return;
    }
    return;
}



//testing functions
void rkCalcComparison(){
    int n = 5000;

    //parameter setup

    elements<double> *curPos = new elements<double>[n];
    elements<double> *hostCurPos = new elements<double>[n];
    for(int i = 0; i < n; i++){
        curPos[i].r = static_cast<double>(rand()%1001)/1000.0 + 0.5;
        curPos[i].theta = static_cast<double>(rand()%40000)/1000.0 - 20.0;
        curPos[i].z = static_cast<double>(rand()%200001)/10000000.0 - 0.001;
        curPos[i].vr = 0.000000018;
        curPos[i].vtheta = 0.00000021;
        curPos[i].vz = -0.000000002;

        hostCurPos[i] = curPos[i];
    }



    coefficients<double> testCoeff;
    for(int i = 0; i < testCoeff.gammaSize; i++){
        testCoeff.gamma[i] = 10;
    }
    for(int i = 0; i < testCoeff.tauSize; i++){
        testCoeff.tau[i] = 10;
    }
    for(int i = 0; i < testCoeff.coastSize; i++){
        testCoeff.coast[i] = 0.5;
    }
    testCoeff.coastThreshold = 0.05;

    double curTime = 1.0;
    double timeFinal = 2.0;
    double stepSize = 4500.0;
    double accel = 5.0e-16;
    elements<double> *v = new elements<double>[n];
    elements<double> k1, k2, k3, k4, k5, k6, k7;


    double *devCurTime;
    double *devTimeFinal;
    double *devStepSize;
    double *devAccel;
    int *devN;
    elements<double> *devV;
    elements<double> *devCurPos;
    coefficients<double> *devTestCoeff;

    hipMalloc((void**) &devCurTime, sizeof(double));
    hipMalloc((void**) &devTimeFinal, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAccel, sizeof(double));
    hipMalloc((void**) &devN, sizeof(int));
    hipMalloc((void**) &devV, n * sizeof(elements<double>));
    hipMalloc((void**) &devCurPos, n * sizeof(elements<double>));
    hipMalloc((void**) &devTestCoeff, sizeof(coefficients<double>));

    hipMemcpy(devCurTime, &curTime, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devN, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devTimeFinal, &timeFinal, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAccel, &accel, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCurPos, curPos, n * sizeof(elements<double>), hipMemcpyHostToDevice);
    hipMemcpy(devTestCoeff, &testCoeff, sizeof(coefficients<double>), hipMemcpyHostToDevice);
    std::cout << "memCpy on" << std::endl;
    rkCalcTest<<<n,1>>>(devCurTime, devTimeFinal, devStepSize, devTestCoeff, devAccel, devV, devCurPos, devN);
    std::cout << "kernel call" << std::endl;
    std::cout << curTime << std::endl;
    std::cout << timeFinal << std::endl;
    std::cout << stepSize << std::endl;
    std::cout << accel << std::endl;
    hipMemcpy(v, devV, n * sizeof(elements<double>), hipMemcpyDeviceToHost);
    std::cout << "memCpy off #1" << std::endl;
    hipMemcpy(curPos, devCurPos, n * sizeof(elements<double>), hipMemcpyDeviceToHost); 
    std::cout << "memCpy off #2" << std::endl;

    elements<double> *hostV = new elements<double>[n];

    std::cout << curTime << std::endl;
    std::cout << timeFinal << std::endl;
    std::cout << stepSize << std::endl;
    std::cout << accel << std::endl;
    //std::cout << testCoeff << std::endl;

    for(int i = 0; i < n; i++){
        std::cout << "i: " << i << std::endl;
        std::cout << hostV[i] << std::endl;
        std::cout << hostCurPos[i] << std::endl;

        rkCalc(curTime, timeFinal, stepSize, hostCurPos[i], testCoeff, accel, hostV[i], k1, k2, k3, k4, k5, k6, k7);
    }

    double errorTol = 0.01;
    for(int i = 0; i < n; i++){
        if(abs(v[i].r - hostV[i].r) > errorTol){
            std::cout << "Thread: " << i << std::endl;
            std::cout << "GPU v: " << v[i] << std::endl;
            std::cout << "CPU v: " << hostV[i] << std::endl;
            std::cout << "difference: " << v[i] - hostV[i] << std::endl;
            std::cout << "GPU curPos: " << curPos[i] << std::endl;
            std::cout << "CPU curPos: " << hostCurPos[i] << std::endl;
            std::cout << "difference: " << curPos[i] - hostCurPos[i] << std::endl;
        }
    }
    std::cout << "done checking for errors" <<std::endl;

    hipFree(devCurTime);
    hipFree(devTimeFinal);
    hipFree(devStepSize);
    hipFree(devAccel);
    hipFree(devN);
    hipFree(devV);
    hipFree(devCurPos);
    hipFree(devTestCoeff);

    delete [] curPos;
    delete [] hostCurPos;
    delete [] hostV;
    delete [] v;
}

__global__ void rkCalcTest(double *curTime, double *timeFinal, double *stepSize, coefficients<double> *testCoeff, double *accel, elements<double> *v, elements<double> *curPos, int *n){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < *n){
        elements<double> k1, k2, k3, k4, k5, k6, k7;
        rkCalc(*curTime, *timeFinal, *stepSize, curPos[threadId], *testCoeff, *accel, v[threadId], k1, k2, k3, k4, k5, k6, k7);
    }
}