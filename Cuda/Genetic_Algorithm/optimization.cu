//Didymos-Optimization_Project:
//Last Editor: Ben
//Tasks Completed: 
    //Put for loop in main to call new optimize() function

#include "../constants.h" //used for wetMass
#include "../Earth_calculations/orbitalMotion.h" //used for trajectory() and trajectoryPrint()
#include "../Earth_calculations/earthInfo.h"
#include "../Runge_Kutta/runge_kuttaCUDA.cuh" //for testing rk4simple

#include <iostream> // cout
#include <iomanip> //used for setw(), sets spaces between values output
#include <time.h> //for seeding the random number generator
#include <random>
#include <chrono>


int main () {
    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device Number: 0 \n";
    std::cout << "- Device name: " << prop.name << std::endl;
    hipSetDevice(0);
    
    // pre-calculate a table of Earth's position within possible mission time range
    //----------------------------------------------------------------
    // Define variables to be passed into EarthInfo
    double startTime = 15778800; // 0.5 year (s)
    double endTime = 78894000; // 2.5 years (s)
    double timeRes = 3600; // (s) position of earth is calculated for every hour

    launchCon = new EarthInfo(startTime, endTime, timeRes); // a global variable to hold Earth's position over time
    //----------------------------------------------------------------

    int blockThreads = 32;
    int numThreads = 2880; // the number of cores on a Tesla k40
    //int numThreads = 1920; // 384 cores on K620 * 5 = 1920

    //std::ofstream efficiencyGraph; // for viewing how many runge-kuttas ran per second for each combination of threads per block and total threads 
    //efficiencyGraph.open("efficiencyGraph.csv");
    double newC3Energy = C3Energy;
    std::ofstream c3EnergyFile;
    c3EnergyFile.open("C3EnergyChange.csv");
    c3EnergyFile << "Number of convergences" << "," << "C3Energy Num" << "," << "\n";
    for(int x = 0; x<30; x++) {
        int convergeNum = 0;
        for(int i = 0; i<10; i++) {
            std::cout << std::endl << "running optimize() with " << blockThreads << " threads per block and " << numThreads << " total threads" << std::endl;
            std::cout << "C3Energy: " << newC3Energy << std::endl;

            optimize(numThreads, blockThreads, newC3Energy, (x*100) + i); // optimize() currently declared in runge_kuttaCUDA.cuh
            if(getConvgFlag()) {
                convergeNum++;
            } else {
                std::cout << "didn't converge" << std::endl;
            }
        }
        c3EnergyFile << convergeNum << "," << newC3Energy << "\n";
        newC3Energy = newC3Energy - 1000;
    }
    
    c3EnergyFile.close();
    delete launchCon;

    return 0;
}
