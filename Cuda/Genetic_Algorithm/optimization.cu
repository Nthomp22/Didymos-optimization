//Didymos-Optimization_Project:
//Last Editor: Ben
//Tasks Completed: 
    //Put for loop in main to call new optimize() function

#include "../constants.h" //used for wetMass
#include "../Earth_calculations/orbitalMotion.h" //used for trajectory() and trajectoryPrint()
#include "../Earth_calculations/earthInfo.h"
#include "../Runge_Kutta/runge_kuttaCUDA.cuh" //for testing rk4simple

#include <iostream> // cout
#include <iomanip> //used for setw(), sets spaces between values output
#include <time.h> //for seeding the random number generator
#include <random>
#include <chrono>


int main () {
    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device Number: 0 \n";
    std::cout << "- Device name: " << prop.name << std::endl;
    hipSetDevice(0);
    
    // pre-calculate a table of Earth's position within possible mission time range
    //----------------------------------------------------------------
    // Define variables to be passed into EarthInfo
    double startTime = 15778800; // 0.5 year (s)
    double endTime = 78894000; // 2.5 years (s)
    double timeRes = 3600; // (s) position of earth is calculated for every hour

    launchCon = new EarthInfo(startTime, endTime, timeRes); // a global variable to hold Earth's position over time
    //----------------------------------------------------------------

    int blockThreads = 32;
    int numThreads = 2880; // the number of cores on a Tesla k40
    //int numThreads = 1920; // 384 cores on K620 * 5 = 1920

    // Initialize the type of thruster to be used
    thruster<double> thrust(0);

    //std::ofstream efficiencyGraph; // for viewing how many runge-kuttas ran per second for each combination of threads per block and total threads 
    //efficiencyGraph.open("efficiencyGraph.csv");
    std::cout << std::endl << "running optimize() with " << blockThreads << " threads per block and " << numThreads << " total threads";
    if (thrust.type) {
        std::cout << " using a NEXT-C thruster" << std::endl;
    }
    else {
        std::cout << " using no thruster" << std::endl;
    }

    optimize(numThreads, blockThreads, thrust); // optimize() currently declared in runge_kuttaCUDA.cuh
    //efficiencyGraph << blockThreads << "," << numThreads << "," << calcPerS  << "\n";
    //efficiencyGraph.close();
    
    delete launchCon;

    return 0;
}
