#include "hip/hip_runtime.h"
// Didymos Optimization Project using CUDA and a genetic algorithm

#include "../constants.h" //used for wetMass
#include "../Earth_calculations/orbitalMotion.h" //used for trajectory() and trajectoryPrint()
#include "../Earth_calculations/earthInfo.h"
#include "../Runge_Kutta/runge_kuttaCUDA.cuh" //for testing rk4simple

#include <iostream> // cout
#include <iomanip> //used for setw(), sets spaces between values output
#include <time.h> //for seeding the random number generator
#include <random>
#include <chrono>
#include <climits> // for INT_MAX

bool changeInBest(double previousBest, double currentBest) { // used to see if the best individual is changing
    if (previousBest == currentBest) {
        return false;
    }
    else {
        return true;
    }
}

// Input: Two ofstreams (one to .csv file and another to binary), current generation number, best individual, and annealing value derived to be used in next generation crossover/mutation
// Output: The two streams are appended the individual's information and anneal
void writeCurrentBestToFile(std::ofstream& ExcelOutput, std::ofstream& BinOutput, unsigned int &currentGeneration, Individual &individual, double& annealing ) {
    // Output the information to excel spreadsheet
    ExcelOutput << currentGeneration << ','
                << individual.posDiff << ','
                << individual.velDiff << ','
                << individual.finalPos.r << ','
                << individual.finalPos.theta << ','
                << individual.finalPos.z << ','
                << individual.finalPos.vr << ','
                << individual.finalPos.vtheta << ','
                << individual.finalPos.vz << ','
                << individual.startParams.y0.r << ','
                << individual.startParams.y0.theta << ','
                << individual.startParams.y0.z << ','
                << individual.startParams.y0.vr << ','
                << individual.startParams.y0.vtheta << ','
                << individual.startParams.y0.vz << ','
                << individual.startParams.alpha << ','
                << individual.startParams.beta << ','
                << individual.startParams.zeta << ','
                << annealing << ","
                << individual.startParams.tripTime
                << std::endl;
    
    // Output the information to binary file for use in the MATLAB code, line breaks and spaces added to help with readibility
    BinOutput.write( (char*)& currentGeneration, sizeof(unsigned int));
 
    BinOutput.write( (char*)& individual.posDiff, sizeof(double));
    BinOutput.write( (char*)& individual.velDiff, sizeof(double));
  
    BinOutput.write( (char*)& individual.finalPos.r,            sizeof(double));
    BinOutput.write( (char*)& individual.finalPos.theta,        sizeof(double));
    BinOutput.write( (char*)& individual.finalPos.z,            sizeof(double));
    BinOutput.write( (char*)& individual.finalPos.vr,           sizeof(double));
    BinOutput.write( (char*)& individual.finalPos.vtheta,       sizeof(double));
    BinOutput.write( (char*)& individual.finalPos.vz,           sizeof(double));
    BinOutput.write( (char*)& individual.startParams.y0.r,      sizeof(double));
    BinOutput.write( (char*)& individual.startParams.y0.theta,  sizeof(double));
    BinOutput.write( (char*)& individual.startParams.y0.z,      sizeof(double));
    BinOutput.write( (char*)& individual.startParams.y0.vr,     sizeof(double));
    BinOutput.write( (char*)& individual.startParams.y0.vtheta, sizeof(double));
    BinOutput.write( (char*)& individual.startParams.y0.vz,     sizeof(double));
    BinOutput.write( (char*)& individual.startParams.alpha,  sizeof(double));
    BinOutput.write( (char*)& individual.startParams.beta,   sizeof(double));
    BinOutput.write( (char*)& individual.startParams.zeta,   sizeof(double));

    BinOutput.write((char*)& annealing, sizeof(double));

    BinOutput.write((char*)& individual.startParams.tripTime, sizeof(double));
}

double optimize(const int numThreads, const int blockThreads) {
    double calcPerS = 0;
    time_t timeSeed = time(0); // Current set to 0 instead of time(0) to ideally help in testing the algorithm
    std::cout << "Seed for this run: " << timeSeed << std::endl; // note there are other mt_rands in the code that use different seeds
    std::cout << "------------------------------------------------------------------------" << std::endl;
    std::mt19937_64 mt_rand(timeSeed);

    // input parameters for rk4Simple which are the same for each thread
    double timeInitial = 0; // the starting time of the trip is always defined as zero   
    double absTol = RK_TOL; // the tolerance is a constant number that is shared amongst all runs
    double stepSize = (orbitalPeriod - timeInitial) / MAX_NUMSTEPS; // the starting step size- same for each run- note that the current step size varies throughout each run

    double annealMax = ANNEAL_MAX;
    double annealMin = ANNEAL_MIN;


    Individual *inputParameters = new Individual[numThreads]; // contains all input parameters besides those which are always common amongst every thread

    const int numStarts = 14; // the number of different sets of starting parameters in the input file

    std::ifstream starts;
    starts.open("../optimizedVector.bin", std::ifstream::in|std::ios::binary); // a file containing the final parameters of converged results from CPU calculations

    double startDoubles;
    
    double previousBest = 0; // set to zero to ensure there is a difference between previousBest and currentBest on generation zero (see changeInBest function)

    // sort the data into 2 dimensions
    // one row is one set of starting parameters
    // each column is a specific variable:
    //    0-6 gamma
    //    7-9 tau
    //    10-12 launch angles
    //    13 trip time
    //    14-19 coast
    double arrayCPU[numStarts][OPTIM_VARS];
    for (int i = 0; i < OPTIM_VARS; i++) { // rows
        for (int j = 0; j < numStarts; j++) { // columns
            starts.read( reinterpret_cast<char*>( &startDoubles ), sizeof startDoubles );
            arrayCPU[j][i] = startDoubles;
        }
    }
    starts.close();

     // set every thread's input parameters to a set of final values from CPU calculations for use as a good starting point
    for (int i = 0; i < numThreads; i++) {
        int row = mt_rand() % numStarts;

        double tripTime = arrayCPU[row][13];

        double alpha = arrayCPU[row][10];

        double beta = arrayCPU[row][11];

        double zeta = arrayCPU[row][12];

        coefficients<double> testcoeff;
        for (int j = 0; j < testcoeff.gammaSize; j++) {
            testcoeff.gamma[j] = arrayCPU[row][j];
        }

        for (int j = 0; j < testcoeff.tauSize; j++) {
            testcoeff.tau[j] =  arrayCPU[row][j+7];
        }

        for (int j = 0; j < testcoeff.coastSize; j++) {
            testcoeff.coast[j] = arrayCPU[row][j+14];
        }

        rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

        inputParameters[i].startParams = example;
    }

    // set every thread's input parameters to random values within a reasonable range
    /*
    for(int i = 0; i < numThreads; i++){ 
        double tripTime = 365*24*3600*(std::rand() % 10001 / 10000.0 + 1.0);
        double alpha = (mt_rand() % 629) / 100.0 - 3.14;
        double beta = (mt_rand() % 629) / 100.0 - 3.14;
        double zeta = (mt_rand() % 315) / 100.0 - 1.57;

        coefficients<double> testcoeff;
        for(int j = 0; j < testcoeff.gammaSize; j++){
            testcoeff.gamma[j] = mt_rand() % 201/10.0 - 10.0;
        }
        for(int j = 0; j < testcoeff.tauSize; j++){
            testcoeff.tau[j] = mt_rand() % 201/10.0 - 10.0;
        }
        for(int j = 0; j < testcoeff.coastSize; j++){
            testcoeff.coast[j] = mt_rand() % 201/10.0 - 10.0;
        }
    
        rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

        inputParameters[i].startParams = example;
    }
    */


    Individual *survivors = new Individual[SURVIVOR_COUNT]; // stores the winners of the head-to-head competition
    int newInd = numThreads; // the whole population is new the first time through the loop

    // setup output of generation results over time onto a .csv file
    std::ofstream generationPerformanceExcel;
    generationPerformanceExcel.open("BestInGenerations.csv");
    // Set first row in the file be a header for the columns
    generationPerformanceExcel << "Gen #" << "," << "posDiff" << "," << "velDiff" << "," 
                               << "rFinal" << "," << "thetaFinal" << "," << "zFinal" << "," << "vrFinal" << "," << "vthetaFinal" << "," << "vzFinal" << ","
                               << "rInitial" << "," << "thetaInitial" << "," << "zInitial" << ","<< "vrInitial" << "," << "vthetaInitial" << "," << "vzInitial" << ","
                               << "alpha" << "," << "beta" << "," << "zeta" << "," << "anneal" << "," << "tripTime" << "\n";

    // setup output of generation results over time onto a .bin file
    std::ofstream generationPerformanceBin("BestInGenerations.bin", std::ios::binary);

    unsigned int generation = 0;    // A counter for number of generations calculated
    
    // A do-while loop that continues until it is determined that the pool of inputParameters has reached desired tolerance level
    
    double currentDistance; // Contains value for how far away the best individual is from the tolerance value
    double tolerance = POSITION_THRESH; // Tolerance for what is an acceptable solution (currently just POSITION_THRESH which is furthest distance from the target allowed)

    do { // Set as a do while loop so that the algorithm is set to run atleast once
        // initialize positions for the new individuals starting at the index of the first new one and going to the end of the array
        initializePosition(inputParameters + (numThreads - newInd), newInd);

        callRK(newInd, blockThreads, inputParameters + (numThreads - newInd), timeInitial, stepSize, absTol, calcPerS); // calculate trajectories for new individuals

        // if we got bad results reset the Individual to random starting values (it may still be used for crossover) and set the final position to be way off so it gets replaced by a new Individual
        for (int k = 0; k < numThreads; k++) { 
            if (isnan(inputParameters[k].finalPos.r) || isnan(inputParameters[k].finalPos.theta) || isnan(inputParameters[k].finalPos.z) 
                 || isnan(inputParameters[k].finalPos.vr) || isnan(inputParameters[k].finalPos.vtheta) || isnan(inputParameters[k].finalPos.vz)){
                
                std::cout << std::endl << std::endl << "NAN FOUND" << std::endl << std::endl;

                double tripTime = 365*24*3600*(std::rand() % 10001 / 10000.0 + 1.0);
                double alpha = (mt_rand() % 629) / 100.0 - 3.14;
                double beta = (mt_rand() % 629) / 100.0 - 3.14;
                double zeta = (mt_rand() % 315) / 100.0 - 1.57;
        
                coefficients<double> testcoeff;
                for (int j = 0; j < testcoeff.gammaSize; j++) {
                    testcoeff.gamma[j] = mt_rand() % 201/10.0 - 10.0;
                }
                for (int j = 0; j < testcoeff.tauSize; j++) {
                    testcoeff.tau[j] = mt_rand() % 201/10.0 - 10.0;
                }
                for (int j = 0; j < testcoeff.coastSize; j++) {
                    testcoeff.coast[j] = mt_rand() % 201/10.0 - 10.0;
                }
            
                rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 
        
                inputParameters[k].startParams = example;

                inputParameters[k].posDiff = 1.0e10;
                inputParameters[k].velDiff = 0.0;
             }
        }

        std::shuffle(inputParameters, inputParameters + numThreads, mt_rand); // shuffle the Individiuals to use random members for the competition
        selectWinners(inputParameters, SURVIVOR_COUNT, survivors); // Choose which individuals are in survivors, not necessarrily only the best ones
        std::sort(inputParameters, inputParameters + numThreads, betterInd); // put the individuals in order so we can replace the worst ones

        // Display a '.' to the terminal to show that a generation has been calculated and sorted
        // This also serves to visually seperate the generation display on the terminal screen
        std::cout << '.';

        
        // Calculate how far the pool is from the ideal cost value (0)
        currentDistance = inputParameters[0].posDiff; // Change this later to take into account more than just the best individual

        // the annealing rate passed in is scaled between ANNEAL_MAX and ANNEAL_MIN, dependent on the ratio between the tolerance and current distance from the tolerance
        // annealMax and annealMin change from the initial ANNEAL_MAX and ANNEAL_MIN whenever CHANGE_CHECK many generations pass without changing the best individual
        double new_anneal =  annealMax - tolerance / currentDistance * (annealMax - annealMin);

        // Display and print Individuals' pos and vel difference every 200 generations to terminal
        if (generation % DISP_FREQ == 0) { 
            // Display the best and worst Individuals in this generation
            std::cout << '\n';
            std::cout << "generation: " << generation << std::endl;
            std::cout << "best:" << std::endl;
            std::cout << "\tposDiff: " << inputParameters[0].posDiff << std::endl;
            std::cout << "\tvelDiff: " << inputParameters[0].velDiff << std::endl;
            std::cout << "\tcost: "    << inputParameters[0].getCost() << std::endl;
            std::cout << "worst:" << std::endl;
            std::cout << "\tposDiff: " << inputParameters[numThreads - 1].posDiff << std::endl;
            std::cout << "\tvelDiff: " << inputParameters[numThreads - 1].velDiff << std::endl;
            std::cout << "\tcost: "    << inputParameters[numThreads - 1].getCost() << std::endl;
            
            
        }
        
        double currentBest;
        if (generation % CHANGE_CHECK == 0) { // Compare current best individual to that from CHANGE_CHECK many generations ago. If they are the same, change size of mutations
            currentBest = inputParameters[0].posDiff;
            if ( !(changeInBest(previousBest, currentBest)) ) { // previousBest starts at 0 to ensure changeInBest = true on generation 0
                annealMax = annealMax*ANNEAL_FACTOR;
                annealMin = annealMin*ANNEAL_FACTOR;
            }
            previousBest = inputParameters[0].posDiff;
        }


        // Write the best and worst Individuals in every 1,000 generations into the files to view progress over generations
        if (generation % WRITE_FREQ == 0) {
            writeCurrentBestToFile(generationPerformanceExcel, generationPerformanceBin, generation, inputParameters[0], new_anneal);
        }

        // Create a new generation
        newInd = crossover(survivors, inputParameters, SURVIVOR_COUNT, numThreads, new_anneal);
        ++generation;
        // If the current distance is still higher than the tolerance we find acceptable, perform the loop again
    } while (currentDistance > tolerance);


    
    // output the best Individuals of the final generation, using writeTrajectoryToFile()
    // Files outputted allows plotting of solutions in matlab
    double *start = new double[OPTIM_VARS];
    double cost = 0;

    // Output to excel
    double annealPlacement = 0; //setting anneal to be a placeholder value that has no real meaning
    writeCurrentBestToFile(generationPerformanceExcel, generationPerformanceBin, generation, inputParameters[0], annealPlacement);

    for (int i = 0; i < BEST_COUNT; i++) {
        for (int j = 0; j < inputParameters[i].startParams.coeff.gammaSize; j++) {
            start[GAMMA_OFFSET + j] = inputParameters[i].startParams.coeff.gamma[j];
        }
        for (int j = 0; j < inputParameters[i].startParams.coeff.tauSize; j++) {
            start[TAU_OFFSET + j] = inputParameters[i].startParams.coeff.tau[j];
        }
        for (int j = 0; j < inputParameters[i].startParams.coeff.coastSize; j++) {
            start[COAST_OFFSET + j] = inputParameters[i].startParams.coeff.coast[j];
        }

        start[TRIPTIME_OFFSET] = inputParameters[i].startParams.tripTime;
        start[ALPHA_OFFSET] = inputParameters[i].startParams.alpha;
        start[BETA_OFFSET] = inputParameters[i].startParams.beta;
        start[ZETA_OFFSET] = inputParameters[i].startParams.zeta;

        cost = inputParameters[i].posDiff; // just look at position difference here for now
        // could instead use a ratio between position and velocity differnce as done in comparison of Individuals
        writeTrajectoryToFile(start, cost, i + 1);
    }


    generationPerformanceExcel.close();
    generationPerformanceBin.close();

    delete [] inputParameters;
    delete [] survivors;

    return calcPerS;
}

int main () {
    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "Device Number: 0 \n";
    std::cout << "- Device name: " << prop.name << std::endl;
    hipSetDevice(0);
    
    // pre-calculate a table of Earth's position within possible mission time range
    //----------------------------------------------------------------
    // Define variables to be passed into EarthInfo
    double startTime = 15778800; // 0.5 year (s)
    double endTime = 78894000; // 2.5 years (s)
    double timeRes = 3600; // (s) position of earth is calculated for every hour

    launchCon = new EarthInfo(startTime, endTime, timeRes); // a global variable to hold Earth's position over time
    //----------------------------------------------------------------

    int blockThreads = 32;
    int numThreads = 2880; // the number of cores on a Tesla k40
    //int numThreads = 1920; // 384 cores on K620 * 5 = 1920

    //std::ofstream efficiencyGraph; // for viewing how many runge-kuttas ran per second for each combination of threads per block and total threads 
    //efficiencyGraph.open("efficiencyGraph.csv");
    std::cout << std::endl << "running optimize() with " << blockThreads << " threads per block and " << numThreads << " total threads" << std::endl;
    
    optimize(numThreads, blockThreads);

    //efficiencyGraph << blockThreads << "," << numThreads << "," << calcPerS  << "\n";
    //efficiencyGraph.close();
    
    delete launchCon;

    return 0;
}
